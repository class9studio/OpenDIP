#include "hip/hip_runtime.h"
#include "cudahead.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>

namespace opendip
{
    __global__ void unique_idx_calc_threadIdx(int * input)
    {
        int tid = threadIdx.x;
        printf("threadIdx : %d, value : %d \n", tid, input[tid]);
    }

    __global__ void unique_gid_calculation(int * input)
    {
        int tid = threadIdx.x;
        int offset = blockIdx.x * blockDim.x;
        int gid = tid + offset;
        printf("blockIdx.x : %d, threadIdx.x : %d, gid : %d, value : %d \n",
            blockIdx.x, tid, gid, input[gid]);
    }

    int unique_gid_calc(void)
    {
    	int array_size = 16;
    	int array_byte_size = sizeof(int) * array_size;
    	int h_data[] = {23,9,4,53,65,12,1,33,87,45,23,12,342,56,44,99};
    
    	for (int i = 0; i < array_size; i++)
    	{
    		printf("%d ", h_data[i]);
    	}
    	printf("\n \n");
    
    	int * d_data;
    	hipMalloc((void**)&d_data, array_byte_size);
    	hipMemcpy(d_data, h_data, array_byte_size, hipMemcpyHostToDevice);
    
    	dim3 block(4);
    	dim3 grid(4);
    	
    	unique_gid_calculation <<< grid, block >> > (d_data);
    	hipDeviceSynchronize();
    
    	hipDeviceReset();
    	return 0;
    }

}