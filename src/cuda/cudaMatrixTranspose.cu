#include "hip/hip_runtime.h"
#include "cudahead.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include ""

#include "common.h"

namespace opendip
{
__global__ void copy_row(int * mat, int * transpose, int nx, int ny)
{
	int ix = blockIdx.x * blockDim.x + threadIdx.x;
	int iy = blockIdx.y * blockDim.y + threadIdx.y;

	if (ix < nx && iy < ny)
	{
		transpose[iy * nx + ix] = mat[iy * nx + ix];
	}
}

__global__ void copy_column(int * mat, int * transpose, int nx, int ny)
{
	int ix = blockIdx.x * blockDim.x + threadIdx.x;
	int iy = blockIdx.y * blockDim.y + threadIdx.y;

	if (ix < nx && iy < ny)
	{
		transpose[ix * ny + iy] = mat[ix * ny + iy];
	}
}

__global__ void transpose_read_row_write_column(int * mat, int * transpose, int nx, int ny)
{
	int ix = blockIdx.x * blockDim.x + threadIdx.x;
	int iy = blockIdx.y * blockDim.y + threadIdx.y;

	if (ix < nx && iy < ny)
	{
		transpose[ix * ny + iy] = mat[iy * nx + ix];
	}
}

__global__ void transpose_read_column_write_row(int * mat, int * transpose, int nx, int ny)
{
	int ix = blockIdx.x * blockDim.x + threadIdx.x;
	int iy = blockIdx.y * blockDim.y + threadIdx.y;

	if (ix < nx && iy < ny)
	{
		transpose[iy * nx + ix] = mat[ix * ny + iy];
	}
}

__global__ void transpose_unroll4_row(int * mat, int * transpose, int nx, int ny)
{
	int ix = blockIdx.x * blockDim.x * 4 + threadIdx.x;
	int iy = blockIdx.y * blockDim.y + threadIdx.y;

	int ti = iy * nx + ix;
	int to = ix * ny + iy;

	if (ix + 3 * blockDim.x < nx && iy < ny)
	{
		transpose[to]						= mat[ti];
		transpose[to + ny*blockDim.x]		= mat[ti + blockDim.x];
		transpose[to + ny * 2 * blockDim.x] = mat[ti + 2 * blockDim.x];
		transpose[to + ny * 3 * blockDim.x] = mat[ti + 3 * blockDim.x];
	}
}

__global__ void transpose_unroll4_col(int * mat, int * transpose, int nx, int ny)
{
	int ix = blockIdx.x * blockDim.x * 4 + threadIdx.x;
	int iy = blockIdx.y * blockDim.y + threadIdx.y;

	int ti = iy * nx + ix;
	int to = ix * ny + iy;

	if (ix + 3 * blockDim.x < nx && iy < ny)
	{
		transpose[ti] = mat[to];
		transpose[ti + blockDim.x] = mat[to + blockDim.x*ny];
		transpose[ti + 2 * blockDim.x] = mat[to + 2 * blockDim.x*ny];
		transpose[ti + 3 * blockDim.x] = mat[to + 3 * blockDim.x*ny];
	}
}

__global__ void transpose_diagonal_row(int * mat, int * transpose, int nx, int ny)
{
	int blk_x = blockIdx.x;
	int blk_y = (blockIdx.x + blockIdx.y) % gridDim.x;

	int ix = blockIdx.x * blk_x + threadIdx.x;
	int iy = blockIdx.y * blk_y + threadIdx.y;

	if (ix < nx && iy < ny)
	{
		transpose[ix * ny + iy] = mat[iy * nx + ix];
	}
}


int matrix_transpose(int kernel_type)
{
	//default values for variabless
	int nx = 1024;
	int ny = 1024;
	int block_x = 128;
	int block_y = 8;
	int kernel_num = 0;


	kernel_num = kernel_type;

	int size = nx * ny;
	int byte_size = sizeof(int*) * size;

	printf("Matrix transpose for %d X % d matrix with block size %d X %d \n",nx,ny,block_x,block_y);

	int * h_mat_array = (int*)malloc(byte_size);
	int * h_trans_array = (int*)malloc(byte_size);
	int * h_ref = (int*)malloc(byte_size);

	//initialize matrix with integers between one and ten
	initialize(h_mat_array,size ,INIT_ONE_TO_TEN);

	//matirx transpose in CPU
	mat_transpose_cpu(h_mat_array, h_trans_array, nx, ny);

	int * d_mat_array, *d_trans_array;
	
	hipMalloc((void**)&d_mat_array, byte_size);
	hipMalloc((void**)&d_trans_array, byte_size);

	hipMemcpy(d_mat_array, h_mat_array, byte_size, hipMemcpyHostToDevice);

	dim3 blocks(block_x, block_y);
	dim3 grid(nx/block_x, ny/block_y);

	void(*kernel)(int*, int*, int, int);
	char * kernel_name;

	switch (kernel_num)
	{
	case 0:
		kernel = &copy_row;
		kernel_name = "Copy row   ";
		break;
	case 1 :
		kernel = &copy_column;
		kernel_name = "Copy column   ";
		break;
	case 2 :
		kernel = &transpose_read_row_write_column;
		kernel_name = " Read row write column ";
		break;
	case 3:
		kernel = &transpose_read_column_write_row;
		kernel_name = "Read column write row ";
		break;
	case 4:
		kernel = &transpose_unroll4_row;
		kernel_name = "Unroll 4 row ";
		break;
	case 5:
		kernel = &transpose_unroll4_col;
		kernel_name = "Unroll 4 col ";
		break;
	case 6:
		kernel = &transpose_diagonal_row;
		kernel_name = "Diagonal row ";
		break;
	}

	printf(" Launching kernel %s \n",kernel_name);

	clock_t gpu_start, gpu_end;
	gpu_start = clock();

	kernel <<< grid, blocks>> > (d_mat_array, d_trans_array,nx, ny);

	hipDeviceSynchronize();

	gpu_end = clock();
	print_time_using_host_clock(gpu_start, gpu_end);

	//copy the transpose memroy back to cpu
	hipMemcpy(h_ref, d_trans_array, byte_size, hipMemcpyDeviceToHost);

	//compare the CPU and GPU transpose matrix for validity
	compare_arrays(h_ref, h_trans_array, size);

	hipDeviceReset();
	return EXIT_SUCCESS;
}
}