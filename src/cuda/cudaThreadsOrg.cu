#include "hip/hip_runtime.h"
#include "cudahead.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>


namespace opendip
{

__global__ void print_threadIds()
{
	printf("threadIdx.x : %d, threadIdx.y : %d, threadIdx.z : %d \n",
		threadIdx.x,threadIdx.y,threadIdx.z);
}


__global__ void print_details()
{
	printf("blockIdx.x : %d, blockIdx.y : %d, blockIdx.z : %d, blockDim.x : %d, blockDim.y : %d, gridDim.x : %d, gridDim.y :%d \n",
		blockIdx.x, blockIdx.y, blockIdx.z,blockDim.x, blockDim.y, gridDim.x, gridDim.y);
}

int threads_organization(void)
{
	int nx, ny;
	nx = 16;
	ny = 16;

	dim3 block(8,8,8);
	dim3 grid(nx/ block.x, ny/block.y);

	print_threadIds << <grid,block >> > ();
	hipDeviceSynchronize();

	hipDeviceReset();
	return 0;
}

int threads_organization1(void)
{
    int nx, ny;
    nx = 16;
    ny = 16;

    dim3 block(8, 8);
    dim3 grid(nx / block.x, ny / block.y);

    print_details << <grid, block >> > ();
    hipDeviceSynchronize();

    hipDeviceReset();
    return 0;
}

}
