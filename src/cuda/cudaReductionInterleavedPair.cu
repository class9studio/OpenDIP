#include "hip/hip_runtime.h"
#include "cudahead.h"
#include <stdio.h>
#include <stdlib.h>

#include "common.h"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

namespace opendip
{
    #define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
  
    inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
    {
        if (code != hipSuccess)
        {
            fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
            if (abort) exit(code);
        }
    }

    //kernel interleaved pairs
    __global__ void reduction_interleaved_pairs(int * int_array, 
        int * temp_array, int size)
    {
        int tid = threadIdx.x;
        int gid = blockDim.x * blockIdx.x + threadIdx.x;

        if (gid > size)
            return;

        for (int offset = blockDim.x/ 2; offset > 0; offset = offset/2)
        {
            if (tid < offset)
            {
                int_array[gid] += int_array[gid + offset];
            }

            __syncthreads();
        }

        if (tid == 0)
        {
            temp_array[blockIdx.x] = int_array[gid];
        }
    }

    int reduction_intleaved_pairs(void)
    {
        printf("Running parallel reduction with interleaved pairs kernel \n");

        int size = 1 << 27;
        int byte_size = size * sizeof(int);
        int block_size = 128;

        int * h_input, *h_ref;
        h_input = (int*)malloc(byte_size);
        initialize(h_input, size, INIT_RANDOM);

        int cpu_result = reduction_cpu(h_input, size);

        dim3 block(block_size);
        dim3 grid(size / block.x);

        printf("Kernel launch parameters || grid : %d, block : %d \n", grid.x, block.x);

        int temp_array_byte_size = sizeof(int)* grid.x;

        h_ref = (int*)malloc(temp_array_byte_size);

        int * d_input, *d_temp;
        gpuErrchk(hipMalloc((void**)&d_input, byte_size));
        gpuErrchk(hipMalloc((void**)&d_temp, temp_array_byte_size));

        gpuErrchk(hipMemset(d_temp, 0, temp_array_byte_size));
        gpuErrchk(hipMemcpy(d_input, h_input, byte_size,
            hipMemcpyHostToDevice));

        reduction_interleaved_pairs <<< grid, block >>> (d_input, d_temp, size);

        gpuErrchk(hipDeviceSynchronize());
        gpuErrchk(hipMemcpy(h_ref, d_temp, temp_array_byte_size, hipMemcpyDeviceToHost));

        int gpu_result = 0;
        for (int i = 0; i < grid.x; i++)
        {
            gpu_result += h_ref[i];
        }

        compare_results(gpu_result, cpu_result);

        gpuErrchk(hipFree(d_input));
        gpuErrchk(hipFree(d_temp));
        free(h_input);
        free(h_ref);

        hipDeviceReset();
        return 0;
    }
}