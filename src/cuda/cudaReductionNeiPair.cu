#include "hip/hip_runtime.h"
#include "cudahead.h"
#include <stdio.h>
#include <stdlib.h>

#include "common.h"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

namespace opendip
{
    #define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
  
    inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
    {
        if (code != hipSuccess)
        {
            fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
            if (abort) exit(code);
        }
    }
    
    //reduction neighbored pairs kernel
    __global__ void redunction_neighbored_pairs(int * input, 
        int * temp, int size)
    {
        int tid = threadIdx.x;
        int gid = blockDim.x * blockIdx.x + threadIdx.x;

        if (gid > size)
            return;

        for (int offset = 1; offset <= blockDim.x/2; offset *= 2)
        {
            if (tid % (2 * offset) == 0)
            {
                input[gid] += input[gid + offset];
            }

            __syncthreads();
        }

        if (tid == 0)
        {
            temp[blockIdx.x] = input[gid];
        }
    }

    int reduction_neighbor_pairs(void)
    {
    	printf("Running neighbored pairs reduction kernel \n");
    
    	int size = 1 << 27; //128 Mb of data
    	int byte_size = size * sizeof(int);
    	int block_size = 128;
    
    	int * h_input, *h_ref;
    	h_input = (int*)malloc(byte_size);
    
    	initialize(h_input, size, INIT_RANDOM);
    
    	//get the reduction result from cpu
    	int cpu_result = reduction_cpu(h_input,size);
    
    	dim3 block(block_size);
    	dim3 grid(size/ block.x);
    
    	printf("Kernel launch parameters | grid.x : %d, block.x : %d \n",
    		grid.x, block.x);
    
    	int temp_array_byte_size = sizeof(int)* grid.x;
    	h_ref = (int*)malloc(temp_array_byte_size);
    
    	int * d_input, *d_temp;
    
    	gpuErrchk(hipMalloc((void**)&d_input,byte_size));
    	gpuErrchk(hipMalloc((void**)&d_temp, temp_array_byte_size));
    
    	gpuErrchk(hipMemset(d_temp, 0 , temp_array_byte_size));
    	gpuErrchk(hipMemcpy(d_input, h_input, byte_size, 
    		hipMemcpyHostToDevice));
    
    	redunction_neighbored_pairs << <grid, block >> > (d_input,d_temp, size);
    
    	gpuErrchk(hipDeviceSynchronize());
    
    	hipMemcpy(h_ref,d_temp, temp_array_byte_size,
    		hipMemcpyDeviceToHost);
    
    	int gpu_result = 0;
    
    	for (int i = 0; i < grid.x; i++)
    	{
    		gpu_result += h_ref[i];
    	}
    
    	//validity check
    	compare_results(gpu_result, cpu_result);
    
    	gpuErrchk(hipFree(d_temp));
    	gpuErrchk(hipFree(d_input));
    
    	free(h_ref);
    	free(h_input);
    
    	gpuErrchk(hipDeviceReset());
    	return 0;
    }
}