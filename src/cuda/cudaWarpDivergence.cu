#include "hip/hip_runtime.h"
#include "cudahead.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

namespace opendip
{
    __global__ void code_without_divergence()
    {
        int gid = blockIdx.x * blockDim.x + threadIdx.x;

        float a, b;
        a = b = 0;

        int warp_id = gid / 32;

        if (warp_id % 2 == 0)
        {
            a = 100.0;
            b = 50.0;
        }
        else
        {
            a = 200;
            b = 75;
        }
    }

    __global__ void divergence_code()
    {
        int gid = blockIdx.x * blockDim.x + threadIdx.x;

        float a, b;
        a = b = 0;

        if (gid % 2 == 0)
        {
            a = 100.0;
            b = 50.0;
        }
        else
        {
            a = 200;
            b = 75;
        }
    }

    int warp_divergence_test(void)
    {
        printf("\n-----------------------WARP DIVERGENCE EXAMPLE------------------------ \n\n");

        int size = 1 << 22;

        dim3 block_size(128);
        dim3 grid_size((size + block_size.x -1)/ block_size.x);

        code_without_divergence << <grid_size, block_size >> > ();
        hipDeviceSynchronize();

        divergence_code << <grid_size, block_size >> > ();
        hipDeviceSynchronize();

        hipDeviceReset();
        return 0;
    }

}