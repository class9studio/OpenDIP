#include "hip/hip_runtime.h"
#include "cudahead.h"
#include <stdio.h>
#include <stdlib.h>

#include "hip/hip_runtime.h"
#include ""


namespace opendip
{
    __global__ void print_details_of_warps()
    {
        int gid = blockIdx.y * gridDim.x * blockDim.x 
            + blockIdx.x * blockDim.x + threadIdx.x;

        int warp_id = threadIdx.x / 32;

        int gbid = blockIdx.y * gridDim.x + blockIdx.x;

        printf("tid : %d, bid.x : %d, bid.y : %d, gid : %d, warp_id : %d, gbid : %d \n",
            threadIdx.x, blockIdx.x, blockIdx.y, gid, warp_id, gbid);
    }

    int warps_index(void)
    {
        dim3 block_size(42);
        dim3 grid_size(2,2);

        print_details_of_warps << <grid_size,block_size >> > ();
        hipDeviceSynchronize();

        hipDeviceReset();
        return 0;
    }

}