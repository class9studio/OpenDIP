#include "hip/hip_runtime.h"
#include "cudahead.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include ""

#include "common.h"

namespace opendip
{
    #define BDIMX 32
    #define BDIMY 32

    #define IPAD 1

    __global__ void setRowReadColPad(int * out)
    {
        __shared__ int tile[BDIMY][BDIMX + IPAD];

        int idx = threadIdx.y * blockDim.x + threadIdx.x;

        //store to the shared memory
        tile[threadIdx.y][threadIdx.x] = idx;

        //waiting for all the threads in thread block to reach this point
        __syncthreads();

        //load from shared memory
        out[idx] = tile[threadIdx.x][threadIdx.y];
    }

    __global__ void setRowReadColDynPad(int * out)
    {
        extern __shared__ int tile[];

        int row_index = threadIdx.y * (blockDim.x+ IPAD) + threadIdx.x;
        int col_index = threadIdx.x * (blockDim.x + IPAD) + threadIdx.y;

        tile[row_index] = row_index;

        __syncthreads();

        out[row_index] = tile[col_index];
    }

    int shared_memory_padding(void) 
    {
        hipSharedMemConfig pConfig;
        hipDeviceGetSharedMemConfig(&pConfig);
        printf("with Bank Mode:%s ", pConfig == 1 ? "4-Byte" : "8-Byte");

        // set up array size 2048
        int nx = BDIMX;
        int ny = BDIMY;

        size_t nBytes = nx * ny * sizeof(int);

        // execution configuration
        dim3 block(BDIMX, BDIMY);
        dim3 grid(1, 1);
        printf("<<< grid (%d,%d) block (%d,%d)>>>\n", grid.x, grid.y, block.x,
            block.y);

        // allocate device memory
        int *d_C;
        hipMalloc((int**)&d_C, nBytes);
        int *gpuRef = (int *)malloc(nBytes);

        hipMemset(d_C, 0, nBytes);
        setRowReadColPad << <grid, block >> >(d_C);
        hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost);

        hipMemset(d_C, 0, nBytes);
        setRowReadColDynPad << <grid, block, sizeof(int) * ((nx + IPAD)*ny) >> > (d_C);
        hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost);

        // free host and device memory
        hipFree(d_C);
        free(gpuRef);

        // reset device
        hipDeviceReset();
        return EXIT_SUCCESS;
    }

}