#include "hip/hip_runtime.h"
#include<stdio.h>
#include <iostream>
#include <assert.h>
#include <hip/hip_runtime.h>

#include "cudahead.h"
#include ""

namespace opendip{

void init_random_i(int *var, int n)
{
    int i;
    for (i = 0; i < n; i++)
        var[i] = 1;
}

/*****************************************************************************
*   Function name: cudaDeviceTest
*   Description  : 测试是否存在device设备-GPU
*   Parameters   : void
*   Return Value : None
*   Spec         :
*   History:
*
*       1.  Date         : 2020-3-1
*           Author       : YangLin
*           Modification : Created function
*****************************************************************************/
__global__ void kernel() 
{
    printf("hello world");
}
int cudaDeviceTest()
{
    kernel<<<1, 1>>>();

	return 0;
}

/*****************************************************************************
*   Function name: cudaVecAddTest
*   Description  : 加速向量数组加运算
*   Parameters   : N      数组长度
*   Return Value : int    success：0  fail: -1
*   Spec         :
*        通过cuda的加法运算，熟悉cuda程序编程的步骤:
*          1. Identity parallelism， 包括: 划分并行运算操作，分配GPU资源-线程
*          2. Write GPU Kernel
*          3. Setup the Problem: 分配内存，初始化操作等
*          4. Launch the Kernel
*          5. Copy results back from GPU
*   History:
*
*       1.  Date         : 2020-3-4
*           Author       : YangLin
*           Modification : Created function
*****************************************************************************/
__global__ void vevAdd(int N, float *a, float *b, float *c)
{
    // work idex, 在launch kernel的时候指定维度
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < N)
    {
      c[idx] = a[idx] + b[idx];
    }
}

int cudaVecAddTest(int N)
{
  float *a, *b, *c;
  float *devA, *devB, *devC;
  a = (float *)malloc(N*sizeof(float));
  b = (float *)malloc(N*sizeof(float));
  c = (float *)malloc(N*sizeof(float));

  //Allocate memory in GPU Globel Memory
  hipMalloc(&devA, N*sizeof(float));
  hipMalloc(&devB, N*sizeof(float));
  hipMalloc(&devC, N*sizeof(float));

  memset(c, 0, N*sizeof(float));
  for (int i = 0; i < N; i++)
  {
    a[i] = 1.0;
    b[i] = 2.0;
  }

  hipMemcpy(devA, a, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(devB, b, N*sizeof(float), hipMemcpyHostToDevice);

  //Lanunch the GPU Kernel
  vevAdd<<<(N+255)/256, 256 >>>(N, devA, devB, devC); //number of thread blocks, shape of thread blocks

  //copy data back
  hipMemcpy(c, devC, N * sizeof(float), hipMemcpyDeviceToHost);

  for(int i=0;i<N;i++)  
	printf("%f\n",c[i]); 
  
  free(a);
  free(b);
  free(c);
  hipFree(devA);
  hipFree(devB);
  hipFree(devC);

  return 0;
}

//One element per thread, using Global Memeory
//input elements are read several times, not an optimized way
#define THREADS_PER_BLOCK 10
#define BLOCK_SIZE THREADS_PER_BLOCK
#define RADIUS 3
__global__ void stencil(int *in, int *out)
{
  int globIdx = blockIdx.x * blockDim.x + threadIdx.x;
  int value = 0;
  for(int offset = -RADIUS; offset <= RADIUS; offset++)
    value += in[globIdx + offset];
  out[globIdx] = value;
}

__global__ void stencil_share_memory(int *in, int *out)
{
  __shared__ int shared[BLOCK_SIZE + 2 * RADIUS];
  int globIdx = blockIdx.x * blockDim.x + threadIdx.x;
  int locIdx = threadIdx.x + RADIUS;
  shared[locIdx] = in[globIdx];
  if(threadIdx.x < RADIUS)
  {
    shared[locIdx - RADIUS] = in[globIdx - RADIUS];
    shared[locIdx + BLOCK_SIZE] = in[globIdx + BLOCK_SIZE];
  }
  __syncthreads();
  int value = 0;
  for(int offset = -RADIUS; offset <= RADIUS; offset++)
    value += shared[locIdx + offset];
  out[globIdx] = value;
}

int cudaStencilTest(int N)
{
  int *in, *out;
  int *dev_in, *dev_out;
  in = (int *)malloc(N*sizeof(int));
  out = (int *)malloc(N*sizeof(int));

  init_random_i(in, N);

  //Allocate memory in GPU Globel Memory
  hipMalloc(&dev_in,  N*sizeof(int));
  hipMalloc(&dev_out, N*sizeof(int));

  // Copie des valeurs des variables de Host vers Device
  hipMemcpy(dev_in, in, N*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dev_out, out, N*sizeof(int), hipMemcpyHostToDevice);

  //Lanunch the GPU Kernel
  stencil_share_memory <<<N/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(dev_in, dev_out);
  //copy data back
  hipMemcpy(out, dev_out, N*sizeof(int), hipMemcpyDeviceToHost);

  for(int i = 0; i < N; i++)
      printf("%i ---i=%d \n", out[i], i);
  

  free(in);   
  free(out);
  hipFree(dev_in);
  hipFree(dev_out);
  
  return 0;
}

struct image_pixel
{
	unsigned char r;
	unsigned char g;
	unsigned char b;
};

__global__ void Rgb2Gray(image_pixel *d_color_data, unsigned char *d_gray_data, int image_rows, int image_cols)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	if (i < image_cols && j < image_rows) {
		int pixel_index = image_cols * j + i;
		image_pixel pixel = d_color_data[pixel_index];
		d_gray_data[pixel_index] = pixel.r * 0.299 + pixel.g * 0.587 + pixel.b * 0.114;
	}
}

Image cudaOpenDipRGB2Gray(Image &src)
{
	assert(src.c == 3);// src img should be color img
	Image gray(src.w, src.h, 1);
	image_pixel *h_src_data = (image_pixel *)src.data;
	// unsigned char *h_gray_data = (unsigned char *)gray.data;

	image_pixel *d_color_data;
	unsigned char *d_gray_data;
	hipMalloc((void**)&d_color_data, src.h*src.w * sizeof(image_pixel));
	hipMalloc((void**)&d_gray_data, src.h*src.w);

	hipMemcpy(d_color_data, h_src_data, src.h*src.w * sizeof(image_pixel), hipMemcpyHostToDevice);

	dim3 block_dim(16, 16);
	dim3 grad_dim(src.w / block_dim.x + 1, src.h / block_dim.y + 1);

	Rgb2Gray << <grad_dim, block_dim >> > (d_color_data, d_gray_data, src.h, src.w);

	hipMemcpy(gray.data, d_gray_data, src.h*src.w, hipMemcpyDeviceToHost);

	hipFree(d_color_data);
	hipFree(d_gray_data);

	return gray;
}

int getThreadNum()
{
	hipDeviceProp_t prop;//hipDeviceProp_t的一个对象
	int count = 0;//GPU的个数
	hipGetDeviceCount(&count);
	std::cout << "gpu count：" << count << '\n';

	hipGetDeviceProperties(&prop, 0);//第二参数为那个gpu
	std::cout << "Max thread nums：" << prop.maxThreadsPerBlock << std::endl;
	std::cout << "Max Block nums: " << prop.maxGridSize[0] << '\t' << prop.maxGridSize[1] << '\t' << prop.maxGridSize[2] << std::endl;
	return prop.maxThreadsPerBlock;
}

} //namespace opendip