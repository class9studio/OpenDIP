#include "hip/hip_runtime.h"
#include<stdio.h>

#include "cudahead.h"

namespace opendip{

void init_random_i(int *var, int n)
{
    int i;
    for (i = 0; i < n; i++)
        var[i] = 1;
}

/*****************************************************************************
*   Function name: cudaDeviceTest
*   Description  : 测试是否存在device设备-GPU
*   Parameters   : void
*   Return Value : None
*   Spec         :
*   History:
*
*       1.  Date         : 2020-3-1
*           Author       : YangLin
*           Modification : Created function
*****************************************************************************/
__global__ void kernel() 
{
    printf("hello world");
}
int cudaDeviceTest()
{
    kernel<<<1, 1>>>();

	return 0;
}

/*****************************************************************************
*   Function name: cudaVecAddTest
*   Description  : 加速向量数组加运算
*   Parameters   : N      数组长度
*   Return Value : int    success：0  fail: -1
*   Spec         :
*        通过cuda的加法运算，熟悉cuda程序编程的步骤:
*          1. Identity parallelism， 包括: 划分并行运算操作，分配GPU资源-线程
*          2. Write GPU Kernel
*          3. Setup the Problem: 分配内存，初始化操作等
*          4. Launch the Kernel
*          5. Copy results back from GPU
*   History:
*
*       1.  Date         : 2020-3-4
*           Author       : YangLin
*           Modification : Created function
*****************************************************************************/
__global__ void vevAdd(int N, float *a, float *b, float *c)
{
    // work idex, 在launch kernel的时候指定维度
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < N)
    {
      c[idx] = a[idx] + b[idx];
    }
}

int cudaVecAddTest(int N)
{
  float *a, *b, *c;
  float *devA, *devB, *devC;
  a = (float *)malloc(N*sizeof(float));
  b = (float *)malloc(N*sizeof(float));
  c = (float *)malloc(N*sizeof(float));

  //Allocate memory in GPU Globel Memory
  hipMalloc(&devA, N*sizeof(float));
  hipMalloc(&devB, N*sizeof(float));
  hipMalloc(&devC, N*sizeof(float));

  memset(c, 0, N*sizeof(float));
  for (int i = 0; i < N; i++)
  {
    a[i] = 1.0;
    b[i] = 2.0;
  }

  hipMemcpy(devA, a, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(devB, b, N*sizeof(float), hipMemcpyHostToDevice);

  //Lanunch the GPU Kernel
  vevAdd<<<(N+255)/256, 256 >>>(N, devA, devB, devC); //number of thread blocks, shape of thread blocks

  //copy data back
  hipMemcpy(c, devC, N * sizeof(float), hipMemcpyDeviceToHost);

  for(int i=0;i<N;i++)  
	printf("%f\n",c[i]); 
  
  free(a);
  free(b);
  free(c);
  hipFree(devA);
  hipFree(devB);
  hipFree(devC);

  return 0;
}

//One element per thread, using Global Memeory
//input elements are read several times, not an optimized way
#define THREADS_PER_BLOCK 10
#define BLOCK_SIZE THREADS_PER_BLOCK
#define RADIUS 3
__global__ void stencil(int *in, int *out)
{
  int globIdx = blockIdx.x * blockDim.x + threadIdx.x;
  int value = 0;
  for(int offset = -RADIUS; offset <= RADIUS; offset++)
    value += in[globIdx + offset];
  out[globIdx] = value;
}

__global__ void stencil_share_memory(int *in, int *out)
{
  __shared__ int shared[BLOCK_SIZE + 2 * RADIUS];
  int globIdx = blockIdx.x * blockDim.x + threadIdx.x;
  int locIdx = threadIdx.x + RADIUS;
  shared[locIdx] = in[globIdx];
  if(threadIdx.x < RADIUS)
  {
    shared[locIdx - RADIUS] = in[globIdx - RADIUS];
    shared[locIdx + BLOCK_SIZE] = in[globIdx + BLOCK_SIZE];
  }
  __syncthreads();
  int value = 0;
  for(int offset = -RADIUS; offset <= RADIUS; offset++)
    value += shared[locIdx + offset];
  out[globIdx] = value;
}

int cudaStencilTest(int N)
{
  int *in, *out;
  int *dev_in, *dev_out;
  in = (int *)malloc(N*sizeof(int));
  out = (int *)malloc(N*sizeof(int));

  init_random_i(in, N);

  //Allocate memory in GPU Globel Memory
  hipMalloc(&dev_in,  N*sizeof(int));
  hipMalloc(&dev_out, N*sizeof(int));

  // Copie des valeurs des variables de Host vers Device
  hipMemcpy(dev_in, in, N*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dev_out, out, N*sizeof(int), hipMemcpyHostToDevice);

  //Lanunch the GPU Kernel
  stencil_share_memory <<<N/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(dev_in, dev_out);
  //copy data back
  hipMemcpy(out, dev_out, N*sizeof(int), hipMemcpyDeviceToHost);

  for(int i = 0; i < N; i++)
      printf("%i ---i=%d \n", out[i], i);
  

  free(in);   
  free(out);
  hipFree(dev_in);
  hipFree(dev_out);
  
  return 0;
}

}
