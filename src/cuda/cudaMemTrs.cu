#include "hip/hip_runtime.h"
#include "cudahead.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <cstring>
#include <time.h>

namespace opendip
{
    __global__ void mem_trs_test(int * input)
    {
        int gid = blockIdx.x * blockDim.x + threadIdx.x;
        printf("tid : %d , gid : %d, value : %d \n",threadIdx.x,gid,input[gid]);
    }

    __global__ void mem_trs_test2(int * input, int size)
    {
        int gid = blockIdx.x * blockDim.x + threadIdx.x;

        if(gid < size)
            printf("tid : %d , gid : %d, value : %d \n", threadIdx.x, gid, input[gid]);
        else
            printf("array out of bounds \n");
    }

    int mem_transfer_test(void)
    {
        int size = 150;
        int byte_size = size * sizeof(int);

        int * h_input;
        h_input = (int*)malloc(byte_size);

        time_t t;
        srand((unsigned)time(&t));
        for (int i = 0; i < size; i++)
        {
            h_input[i] = (int)(rand() & 0xff);
        }

        int * d_input;
        hipMalloc((void**)&d_input, byte_size);

        hipMemcpy(d_input,h_input,byte_size,hipMemcpyHostToDevice);

        dim3 block(32);
        dim3 grid(5);

        mem_trs_test2 << <grid, block >> > (d_input,size);
        hipDeviceSynchronize();

        hipFree(d_input);
        free(h_input);

        hipDeviceReset();
        return 0;
    }

}