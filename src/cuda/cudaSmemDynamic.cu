#include "hip/hip_runtime.h"
#include "cudahead.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include ""

#include "common.h"

#define BDIMX 32
#define BDIMY 32


namespace opendip
{
    __global__ void setRowReadColDyn(int * out)
    {
        extern __shared__ int tile[];

        int row_index = threadIdx.y * blockDim.x + threadIdx.x;
        int col_index = threadIdx.x * blockDim.y + threadIdx.y;

        tile[row_index] = row_index;

        __syncthreads();

        out[row_index] = tile[col_index];

    }

    int shared_memory_dynamic(void)
    {
        hipSharedMemConfig pConfig;
        hipDeviceGetSharedMemConfig(&pConfig);
        printf("with Bank Mode:%s ", pConfig == 1 ? "4-Byte" : "8-Byte");

        // set up array size 2048
        int nx = BDIMX;
        int ny = BDIMY;

        size_t nBytes = nx * ny * sizeof(int);

        // execution configuration
        dim3 block(BDIMX, BDIMY);
        dim3 grid(1, 1);
        printf("<<< grid (%d,%d) block (%d,%d)>>>\n", grid.x, grid.y, block.x,
            block.y);

        // allocate device memory
        int *d_C;
        hipMalloc((int**)&d_C, nBytes);
        int *gpuRef = (int *)malloc(nBytes);

        hipMemset(d_C, 0, nBytes);
        setRowReadColDyn << <grid, block, sizeof(int) * (nx*ny) >> >(d_C);
        hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost);

        // free host and device memory
        hipFree(d_C);
        free(gpuRef);

        // reset device
        hipDeviceReset();
        return EXIT_SUCCESS;
    }

}